#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

cv::Mat imageRGBA;
cv::Mat imageGrey;

uchar4        *d_rgbaImage__;
unsigned char *d_greyImage__;

size_t numRows() { return imageRGBA.rows; }
size_t numCols() { return imageRGBA.cols; }

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

void preProcess(uchar4 **inputImage, unsigned char **greyImage,
                uchar4 **d_rgbaImage, unsigned char **d_greyImage,
                const std::string &filename) {
  //make sure the context initializes ok
  checkCudaErrors(hipFree(0));

  cv::Mat image;
  image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
  if (image.empty()) {
    std::cerr << "Couldn't open file: " << filename << std::endl;
    exit(1);
  }

  cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

  //allocate memory for the output
  imageGrey.create(image.rows, image.cols, CV_8UC1);

  //This shouldn't ever happen given the way the images are created
  //at least based upon my limited understanding of OpenCV, but better to check
  if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
    std::cerr << "Images aren't continuous!! Exiting." << std::endl;
    exit(1);
  }

  *inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
  *greyImage  = imageGrey.ptr<unsigned char>(0);

  const size_t numPixels = numRows() * numCols();
  //allocate memory on the device for both input and output
  checkCudaErrors(hipMalloc(d_rgbaImage, sizeof(uchar4) * numPixels));
  checkCudaErrors(hipMalloc(d_greyImage, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_greyImage, 0, numPixels * sizeof(unsigned char))); //make sure no memory is left laying around

  //copy input array to the GPU
  checkCudaErrors(hipMemcpy(*d_rgbaImage, *inputImage, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

  d_rgbaImage__ = *d_rgbaImage;
  d_greyImage__ = *d_greyImage;
}

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,unsigned char* const greyImage,int numRows, int numCols){
	int threadId = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  	if (threadId <  numRows * numCols){
    		const unsigned char R = rgbaImage[threadId].x;
    		const unsigned char G = rgbaImage[threadId].y;
    		const unsigned char B = rgbaImage[threadId].z;
    		greyImage[threadId] = .299f * R + .587f * G + .114f * B;
  	}
}

void postProcess(const std::string& output_file, unsigned char* data_ptr) {
  	cv::Mat output(numRows(), numCols(), CV_8UC1, (void*)data_ptr);
  	//output the image
  	cv::imwrite(output_file.c_str(), output);
}

void cleanup(){
  	//cleanup
  	hipFree(d_rgbaImage__);
  	hipFree(d_greyImage__);
}

int main(int argc,char* argv[]){

	//load input file
	std::string input_file = "../files/cinque_terre_small.jpg";
	//define output file
	std::string output_file = "../files/gray.jpg";

	uchar4 *h_rgbaImage, *d_rgbaImage;
	unsigned char *h_greyImage, *d_greyImage;

	//load the image and give us our input and output pointers
  	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

	int thread = 16;
  	int grid = (numRows()*numCols() + thread - 1)/ (thread * thread);
  	const dim3 blockSize(thread, thread);
  	const dim3 gridSize(grid);
  	rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows(), numCols());


	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	size_t numPixels = numRows()*numCols();
  	checkCudaErrors(hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));

	//check results and output the grey image
  	postProcess(output_file, h_greyImage);

	cleanup();
}

