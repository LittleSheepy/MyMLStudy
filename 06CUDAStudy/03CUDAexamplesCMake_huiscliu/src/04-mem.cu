#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>

typedef double FLOAT;
__global__ void sum(FLOAT* x)
{
    int tid = threadIdx.x;
    printf(">>>>>sumt id = %d\n", tid);
    x[tid] += 1;
}

int mem04()
{
    int N = 4;
    int nbytes = N * sizeof(FLOAT);

    FLOAT *dx = NULL, *hx = NULL;
    int i;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);

    if (dx == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    /* alllocate CPU host mem: memory copy is faster than malloc */
    hx = (FLOAT*)malloc(nbytes);

    if (hx == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }

    /* init */
    printf(">>>>>hx original: \n");
    for (i = 0; i < N; i++) {
        hx[i] = i;
        printf("%g\n", hx[i]);
    }
    printf("<<<<<hx original: \n");

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    sum<<<1, N>>>(dx);

    /* let GPU finish */
    hipDeviceSynchronize();

    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);


    printf("\nhx from GPU: \n");
    for (i = 0; i < N; i++) {
        printf("%g\n", hx[i]);
    }

    hipFree(dx);
    free(hx);

    return 0;
}


int mem04_host()
{
    int N = 4;
    int nbytes = N * sizeof(FLOAT);

    FLOAT* dx = NULL, * hx = NULL;
    int i;

    /* allocate GPU mem */
    hipMalloc((void**)&dx, nbytes);

    if (dx == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    /* alllocate CPU host mem: memory copy is faster than malloc */
    //hx = (FLOAT*)malloc(nbytes);
    hipMemAllocHost((void**)&hx, nbytes);

    if (hx == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }

    /* init */
    printf(">>>>>hx original: \n");
    for (i = 0; i < N; i++) {
        hx[i] = i;
        printf("%g\n", hx[i]);
    }
    printf("<<<<<hx original: \n");

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    sum << <1, N >> > (dx);

    /* let GPU finish */
    hipDeviceSynchronize();

    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);

    /* �첽copy�Ļ� �͵ü�hipDeviceSynchronize ͬ��*/
    //hipMemcpyAsync(hx, dx, nbytes, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();


    printf("\nhx from GPU: \n");
    for (i = 0; i < N; i++) {
        printf("%g\n", hx[i]);
    }

    hipFree(dx);
    //free(hx);
    hipHostFree(hx);

    return 0;
}
