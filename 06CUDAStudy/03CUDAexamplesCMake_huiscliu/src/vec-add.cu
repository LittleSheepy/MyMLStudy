#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "reduction_aux.h"

typedef float FLOAT;

/* host, add */
void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N);

/* device function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 1D block */
    int idx = get_tid();

    if (idx < N) z[idx] = z[idx] + y[idx] + x[idx];
}

void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    int i;

    for (i = 0; i < N; i++) z[i] = z[i] + y[i] + x[i];
}

int vec_add()
{
    int N = 20000000;
    int nbytes = N * sizeof(FLOAT);

    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid = dim3(s, s);

    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL, *hy = NULL;
    FLOAT *dz = NULL, *hz = NULL;

    int itr = 30;
    int i;
    double th, td;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    if (dx == NULL || dy == NULL || dz == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %.2f MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);
    hz = (FLOAT *) malloc(nbytes);

    if (hx == NULL || hy == NULL || hz == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %.2f MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
        hz[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dz, hz, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    hipDeviceSynchronize();
    td = get_time();
    
    for (i = 0; i < itr; i++) vec_add<<<grid, bs>>>(dx, dy, dz, N);

    hipDeviceSynchronize();
    td = get_time() - td;

    /* CPU */
    th = get_time();
    for (i = 0; i < itr; i++) vec_add_host(hx, hy, hz, N);
    th = get_time() - th;

    printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);

    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);

    return 0;
}
